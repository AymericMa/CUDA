#include "hip/hip_runtime.h"
/*
This program will numerically compute the integral of
                  4/(1+x*x)
from 0 to 1.  The value of this integral is pi -- which
is great since it gives us an easy way to check the answer.
The is the original sequential program.  It uses the timer
from the OpenMP runtime library
History: Written by Tim Mattson, 11/99.
*/
#include <stdio.h>
#include "hip/hip_runtime.h"
/*#include <omp.h>*/
static long num_steps = 1000000000;
double step;
double pi;


__global__ void calculation (int num_steps,int step,double*tabsum){

	int i;
	double x, sum= 0.0;
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int num_thread = gridDim.x*blockDim.x;

	for (i=tid;i<= num_steps; i+=num_thread){
		x = (i-0.5)*step;
		sum = sum + 4.0/(1.0+x*x);
	}

	__syncthreads();
	*(tabsum + blockIdx.x - 1) = sum;
}


int main ()
{
	  hipEvent_t start_time, stop_time;
		float elapsed_time;
		double *tabsum=0;

		step = 1.0/(double) num_steps;

		hipEventCreate( &start_time );
    hipEventCreate( &stop_time );
    hipEventRecord( start_time, 0 );
/*	  start_time = omp_get_wtime();*/
	  calculation<<<16,16>>>(num_steps,step,*tabsum);

		for (int i=0;i<blockDim.x;i++){

			pi = **(tabsum + i) * step;
		}

		hipEventRecord(stop_time,0);
    hipEventSynchronize( stop_time );
		hipEventElapsedTime(&elapsed_time,start_time,stop_time);
/*	  run_time = omp_get_wtime() - start_time;*/
	  printf("\n pi with %ld steps is %lf in %lf millisecond\n ",num_steps,pi,elapsed_time);
		hipEventDestroy( start_time );
    hipEventDestroy( stop_time );
}
